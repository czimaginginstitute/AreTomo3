#include "hip/hip_runtime.h"
#include "CFindCtfInc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace McAreTomo::AreTomo;
using namespace McAreTomo::AreTomo::FindCtf;

//----------------------------------------------------------
// s_gfCtfParam[0]: wavelength in pixel
// s_gfCtfParam[1]: Cs in pixel
//----------------------------------------------------------
static __device__ __constant__ float s_gfCtfParam[2];

//--------------------------------------------------------------------
// 1. fDfMean, fDfSigma are in pixel, not angstrom.
//--------------------------------------------------------------------
static __device__ float mGCalcPhase
(	float fDfMean,
	float fDfSigma,
	float fAzimuth,
	float fExtPhase,
	float fY
)
{	float fX = blockIdx.x * 0.5f / (gridDim.x - 1);
	//-----------------
	float fS2 = fX * fX + fY * fY;
	float fW2 = s_gfCtfParam[0] * s_gfCtfParam[0];
	//-----------------
	fX = atanf(fY / (fX + (float)1e-30));
	fX = fDfMean + fDfSigma * cosf(2.0f * (fX - fAzimuth));
	//-----------------
	fX = fExtPhase + 3.1415926f * s_gfCtfParam[0] * fS2
	   * (fX - 0.5f * fW2 * fS2 * s_gfCtfParam[1]);
	return fX;
}

//--------------------------------------------------------------------
// 1. Flip the phase of image Fourier transform (gCmp) when CTF is
//    positive. This keeps particles dark.
//--------------------------------------------------------------------
static __global__ void mGPhaseFlip
(	float fDfMean,
	float fDfSigma,
	float fAzmuth,
	float fExtPhase,
	hipfftComplex* gCmp,
	int iCmpY
)
{	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(blockIdx.x == 0 && y == 0) return;
	if(y >= iCmpY) return;
	//-----------------
	float fY = y / (float)iCmpY;
	if(fY > 0.5f) fY = fY - 1.0f;
	//-----------------------------------------------
	// fY is the phase now.
	//-----------------------------------------------
	fY = mGCalcPhase(fDfMean, fDfSigma, 
	   fAzmuth, fExtPhase, fY);
	fY = -sinf(fY);
	if(fY <= 0) return;
	//-----------------
	int i = y * gridDim.x + blockIdx.x;
	gCmp[i].x = -gCmp[i].x;
	gCmp[i].y = -gCmp[i].y;
}

static __global__ void mGWeinerFilter
(	float fDfMean,
	float fDfSigma,
	float fAzmuth,
	float fExtPhase,
	float fBFactor,
	hipfftComplex* gCmp,
	int iCmpY
)
{	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(blockIdx.x == 0 && y == 0) return;
	if(y >= iCmpY) return;
	//-----------------
	float fX = blockIdx.x * 0.5f / (gridDim.x - 1.0f);
	float fY = y / (float)iCmpY;
	if(fY > 0.5f) fY = fY - 1.0f;
	float fR2 = (fX * fX + fY * fY);
	//-----------------
	float fCTF = mGCalcPhase(fDfMean, fDfSigma,
	   fAzmuth, fExtPhase, fY);
	fCTF = -sinf(fCTF);
	//-----------------
	float fSign = (fCTF <= 0) ? 1.0f : -1.0f; // dark particles
	fX = 8.0f * expf(fR2 * 2.0f);
	fCTF = (fabsf(fCTF) + fX) / (fX + 1.0f) * fSign;
	fCTF = expf(-fBFactor * sqrtf(fR2)) / fCTF;
	//-----------------
	int i = y * gridDim.x + blockIdx.x;
	gCmp[i].x *= fCTF;
	gCmp[i].y *= fCTF;
}

/*
static __global__ void mGWeinerFilter
(       float fDfMean,
        float fDfSigma,
        float fAzmuth,
        float fExtPhase,
        float* gfNoise2,
        hipfftComplex* gCmp,
        int iCmpY
)
{       int y = blockIdx.y * blockDim.y + threadIdx.y;
        if(blockIdx.x == 0 && y == 0) return;
        if(y >= iCmpY) return;
	int i = y * gridDim.x + blockIdx.x;
        //-----------------
        float fY = y / (float)iCmpY;
        if(fY > 0.5f) fY = fY - 1.0f;
        //-----------------
        float fCTF = mGCalcPhase(fDfMean, fDfSigma,
           fAzmuth, fExtPhase, fY);
        fCTF = sinf(fCTF);
        //-----------------
        float fFilter = blockIdx.x * 0.5f / (gridDim.x - 1);
        fFilter = expf(-10.0 * (fFilter * fFilter + fY * fY));
	//-----------------
	float fAmp2 = gCmp[i].x * gCmp[i].x + gCmp[i].y * gCmp[i].y;
       	fCTF = fAmp2 * fCTF / (fAmp2 * fCTF * fCTF + gfNoise2[0] + 1.0f);
	fCTF *= fFilter;
        //-----------------
        gCmp[i].x *= fCTF;
        gCmp[i].y *= fCTF;
}
*/
/*
static __global__ void mGCalcNoise2
(	hipfftComplex* gCmp,
	int iCmpY,
	float* gfNoise2
)
{	extern __shared__ float s_afShared[];
	float* s_afCount = &s_afShared[blockDim.y];
	//-----------------
	float fNx = (gridDim.x - 1.0f) * 2.0f;
	float fSumAmp2 = 0.0f;
	float fCount = 0;
	hipfftComplex aCmp;
	int i = 0;
	//-----------------
	for(int y=threadIdx.y; y<iCmpY; y+=blockDim.y)
	{	float fY = y / (float)iCmpY;
		if(fY > 0.5f) fY -= 1.0f;
		for(int x=0; x<gridDim.x; x++)
		{	float fX = x / fNx;
			fX = sqrtf(fX * fX + fY * fY);
			if(fX < 0.5f) continue;
			//---------------
			aCmp = gCmp[y * gridDim.x + blockIdx.x];
			fSumAmp2 += (aCmp.x * aCmp.x + aCmp.y * aCmp.y);
			fCount += 1;
		}
	}
	s_afShared[threadIdx.y] = fSumAmp2;
	s_afCount[threadIdx.y] = fCount;
	__syncthreads();
	//-----------------
	int iOffset = blockDim.y / 2;
	while(iOffset > 0)
	{	if(threadIdx.y < iOffset)
		{	i = iOffset + threadIdx.y;
			s_afShared[threadIdx.y] += s_afShared[i];
			s_afCount[threadIdx.y] += s_afCount[i];
		}
		__syncthreads();
		iOffset /= 2;
	}
	if(threadIdx.y != 0) return;
	//-----------------	
	if(s_afCount[0] == 0) gfNoise2[0] = 0.0f;
	else gfNoise2[0] = s_afShared[0] / s_afCount[0];	
}
*/

GCorrCTF2D::GCorrCTF2D(void)
{
	m_gfNoise2 = 0L;
	m_bPhaseFlip = false;
	m_fBFactor = 15.0f;
}

GCorrCTF2D::~GCorrCTF2D(void)
{
	if(m_gfNoise2 != 0L) hipFree(m_gfNoise2);
	m_gfNoise2 = 0L;
}

void GCorrCTF2D::SetParam(MD::CCtfParam* pCtfParam)
{
	float afCtfParam[2] = {0.0f};
	afCtfParam[0] = pCtfParam->m_fWavelength;
	afCtfParam[1] = pCtfParam->m_fCs;
	hipMemcpyToSymbol(HIP_SYMBOL(s_gfCtfParam), afCtfParam, sizeof(float) * 2);
	//-----------------
	m_fAmpPhase = (float)atanf(pCtfParam->m_fAmpContrast / (1.0f 
	   - pCtfParam->m_fAmpContrast * pCtfParam->m_fAmpContrast));
	//-----------------
	if(m_gfNoise2 == 0L) hipMalloc(&m_gfNoise2, sizeof(float));
}

void GCorrCTF2D::SetPhaseFlip(bool bValue)
{
	m_bPhaseFlip = bValue;
}

void GCorrCTF2D::SetLowpass(int iBFactor)
{
	m_fBFactor = (float)iBFactor;
	if(m_fBFactor < 0) m_fBFactor = 0.0f;
}

void GCorrCTF2D::DoIt
(	float fDfMin,   float fDfMax, 
	float fAzimuth, float fExtPhase,
	float fTilt, hipfftComplex* gCmp, 
	int* piCmpSize, hipStream_t stream
)
{	dim3 aBlockDim(1, 512);
	dim3 aGridDim(piCmpSize[0], 1);
	aGridDim.y = (piCmpSize[1] + aBlockDim.y - 1) / aBlockDim.y;
	//-----------------
	//size_t tSmBytes = sizeof(float) * aBlockDim.y * 2;
	//mGCalcNoise2<<<aGridDim, aBlockDim, tSmBytes, stream>>>(gCmp,
	//   piCmpSize[1], m_gfNoise2);
	//-----------------
	float fDfMean = 0.5f * (fDfMin + fDfMax);
        float fDfSigma = 0.5f * (fDfMax - fDfMin);
	float fAddPhase = m_fAmpPhase + fExtPhase;
	//-----------------
	if(m_bPhaseFlip)
        {	mGPhaseFlip<<<aGridDim, aBlockDim, 0, stream>>>(fDfMean,
		   fDfSigma, fAzimuth, fAddPhase, gCmp, piCmpSize[1]);
	}
	else	
	{	float fBFactor = m_fBFactor / 
		   (float)(cos(fTilt * 0.01745) + 0.001f);
		//----------------
		mGWeinerFilter<<<aGridDim, aBlockDim, 0, stream>>>(fDfMean, 
		   fDfSigma, fAzimuth, fAddPhase, fBFactor, 
		   gCmp, piCmpSize[1]);
	}
}

void GCorrCTF2D::DoIt
(	MD::CCtfParam* pCtfParam, 
	float fTilt,
	hipfftComplex* gCmp, 
	int* piCmpSize,
	hipStream_t stream
)
{	this->SetParam(pCtfParam);
	this->DoIt(pCtfParam->m_fDefocusMin, pCtfParam->m_fDefocusMax,
	   pCtfParam->m_fAstAzimuth, pCtfParam->m_fExtPhase,
	   fTilt, gCmp, piCmpSize);
}
