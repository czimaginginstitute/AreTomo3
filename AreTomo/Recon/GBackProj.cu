#include "hip/hip_runtime.h"
#include "CReconInc.h"
#include <memory.h>
#include <stdio.h>

using namespace McAreTomo::AreTomo::Recon;

// iProjX, iPadProjX, iAllProjs, iVolX
static __device__ __constant__ int giSize[4]; 
//-------------------------------------------
static __global__ void mGBackProj
(	float* gfPadSinogram,
	float* gfCosSin,
	int iStartProj,
	int iNumProjs,
	bool bSart,
	float fRelax,
	float* gfVolXZ
)
{	int iX = blockIdx.x * blockDim.x + threadIdx.x;
	if(iX >= giSize[3]) return;
	//-------------------------
	float fX = iX + 0.5f - giSize[3] * 0.5f;
	float fZ = blockIdx.y + 0.5f - gridDim.y * 0.5f;
	float fProjCentX = giSize[0] / 2.0f;
	int iProjEndX = giSize[0] - 2.0f;
	//-------------------------------
        float fInt = 0.0f;
	int i, iProj = 0, iCount = 0;
	for(i=0; i<iNumProjs; i++)
	{	iProj = i + iStartProj;	
		float fXp = fX * gfCosSin[2 * iProj] 
			+ fZ * gfCosSin[2 * iProj +1] 
			+ fProjCentX;
		if(fXp < 0 || fXp > iProjEndX) continue;
		//--------------------------------------
		fXp = gfPadSinogram[iProj * giSize[1] + (int)fXp];
		if(fXp <= (float)-1e10) continue;
		//-------------------------------
		fInt += fXp;
		iCount += 1;
        }
	if(iCount <= 0) return;
	//---------------------
	i = blockIdx.y * giSize[3] + iX;
	fInt = fRelax * fInt / iCount + gfVolXZ[i];
	//-----------------------------------------
	if(bSart) gfVolXZ[i] = fmaxf(fInt, 0.0f);
	else gfVolXZ[i] = fInt;
}

GBackProj::GBackProj(void)
{
}

GBackProj::~GBackProj(void)
{
}

void GBackProj::SetSize(int* piPadProjSize, int* piVolSize)
{
	int iProjX = (piPadProjSize[0] / 2 - 1) * 2;
	int aiSize[] = {iProjX, piPadProjSize[0], 
	   piPadProjSize[1], piVolSize[0]};
	hipMemcpyToSymbol(HIP_SYMBOL(giSize), aiSize, sizeof(giSize));
	//-------------------------------------------------
	m_aBlockDim.x = 512;
	m_aBlockDim.y = 1;
	m_aGridDim.x = (piVolSize[0] + m_aBlockDim.x - 1) / m_aBlockDim.x;
	m_aGridDim.y = piVolSize[1];
} 

void GBackProj::DoIt
(	float* gfPadSinogram,
	float* gfCosSin,  // cosine and sine of all tilt angles
	int iStartProj,
	int iNumProjs,
	bool bSart,
	float fRelax,
	float* gfVolXZ,
	hipStream_t stream
)
{	mGBackProj<<<m_aGridDim, m_aBlockDim, 0, stream>>>
	( gfPadSinogram, gfCosSin, iStartProj, iNumProjs, 
	  bSart, fRelax, gfVolXZ
	);
}
