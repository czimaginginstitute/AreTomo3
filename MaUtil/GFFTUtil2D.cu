#include "hip/hip_runtime.h"
#include "CMaUtilInc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace McAreTomo::MaUtil;

static __global__ void mGMultiplyFactor
(	hipfftComplex* gCmp,
	int nxy,
	float fFactor
)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= nxy) return;
	gCmp[i].x *= fFactor;
	gCmp[i].y *= fFactor;
}

static __global__ void mGGetAmp
(	hipfftComplex* gCmp,
	int nxy,
	float* gfAmp
)
{	int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= nxy) return;
	gfAmp[i] = sqrtf(gCmp[i].x * gCmp[i].x + gCmp[i].y * gCmp[i].y);
}

static __global__ void mGShiftFrame
(       hipfftComplex* gComp,
        float fShiftX,
        float fShiftY,
        int iCmpX,
        int iCmpY
)
{       int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= iCmpX || y >= iCmpY) return;
        int i = y * iCmpX + x;
	//-----------------
	if(y  > (iCmpY / 2)) y -= iCmpY;
        float fPhaseShift = x * fShiftX + y * fShiftY;
        float fCos = cosf(fPhaseShift);
        float fSin = sinf(fPhaseShift);
	//-----------------
	float fRe = fCos * gComp[i].x - fSin * gComp[i].y;
        float fIm = fCos * gComp[i].y + fSin * gComp[i].x;
        gComp[i].x = fRe;
        gComp[i].y = fIm;
}

static __global__ void mGLowpass
(	hipfftComplex* gInCmp,
	int iCmpY,
	float fBFactor,
	hipfftComplex* gOutCmp
)
{	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y >= iCmpY) return;
	int i = y * gridDim.x + blockIdx.x;
	//-----------------
	if(y > (iCmpY / 2)) y -= iCmpY;
	float fFilt = expf(fBFactor * (blockIdx.x * blockIdx.x + y * y));
	gOutCmp[i].x = gInCmp[i].x * fFilt;
	gOutCmp[i].y = gInCmp[i].y * fFilt;
}


GFFTUtil2D::GFFTUtil2D(void)
{
}

GFFTUtil2D::~GFFTUtil2D(void)
{
}

void GFFTUtil2D::Multiply
( 	hipfftComplex* gComp,
	int* piCmpSize,
	float fFactor,
        hipStream_t stream
)
{	int nxy = piCmpSize[0] * piCmpSize[1];
        dim3 aBlockDim(512, 1, 1);
        dim3 aGridDim((nxy + aBlockDim.x - 1) / aBlockDim.x, 1, 1);
	//----------------------------------
	mGMultiplyFactor<<<aGridDim, aBlockDim, 0, stream>>> (gComp, nxy, fFactor);
}

void GFFTUtil2D::GetAmp
(	hipfftComplex* gComp,
	int* piCmpSize,
	float* pfAmpRes,
	bool bGpuRes,
        hipStream_t stream
)
{	int nxy = piCmpSize[0] * piCmpSize[1];
        dim3 aBlockDim(512, 1, 1);
        dim3 aGridDim((nxy + aBlockDim.x - 1) / aBlockDim.x, 1, 1);
	//------------------------------
	size_t tBytes = sizeof(float) * nxy;
	float* gfAmp = nullptr;
	if(bGpuRes) gfAmp = pfAmpRes;
	else hipMalloc(&gfAmp, tBytes);
	//----------------------------------
	mGGetAmp<<<aGridDim, aBlockDim, 0, stream>>> (gComp, nxy, gfAmp);
	//---------------------------
	if(bGpuRes) return;
	hipMemcpy(pfAmpRes, gfAmp, tBytes, hipMemcpyDeviceToHost);
	hipFree(gfAmp);
}
	
void GFFTUtil2D::Shift
(       hipfftComplex* gComp,
	int* piCmpSize,
        float* pfShift,
        hipStream_t stream
)
{       if(pfShift == 0L) return;
        if(pfShift[0] == 0.0f && pfShift[1] == 0.0f) return;
	//-----------------
	dim3 aBlockDim(128, 4, 1);
        int iGridX = (piCmpSize[0] + aBlockDim.x - 1) / aBlockDim.x;
        int iGridY = (piCmpSize[1] + aBlockDim.y - 1) / aBlockDim.y;
        dim3 aGridDim(iGridX, iGridY);
	//-----------------
	int iNx = 2 * (piCmpSize[0] - 1);
        float f2PI = (float)(8 * atan(1.0));
        float fShiftX = pfShift[0] * f2PI / iNx;
        float fShiftY = pfShift[1] * f2PI / piCmpSize[1];
        //-----------------
	mGShiftFrame<<<aGridDim, aBlockDim, 0, stream>>>
        (  gComp, fShiftX, fShiftY, piCmpSize[0], piCmpSize[1]
        );
}

void GFFTUtil2D::Lowpass
(	hipfftComplex* gInCmp,
	hipfftComplex* gOutCmp,
	int* piCmpSize,
	float fBFactor
)
{	int iNx = (piCmpSize[0] - 1) * 2;
	double dTemp = iNx * iNx + piCmpSize[1] * piCmpSize[1];
	float fScale = (float)(-fBFactor / dTemp);
	//-----------------
	dim3 aBlockDim(1, 512);
	dim3 aGridDim(piCmpSize[0], 1);
	aGridDim.y = piCmpSize[1] / aBlockDim.y + 1;
	mGLowpass<<<aGridDim, aBlockDim>>>(gInCmp, piCmpSize[1], 
	   fScale, gOutCmp);
}

