#include "hip/hip_runtime.h"
#include "CMaUtilInc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace McAreTomo::MaUtil;

static __global__ void mGResize
( 	hipfftComplex* gCmpIn,
	int iCmpSizeInX,
	int iCmpSizeInY,
  	hipfftComplex* gCmpOut, 
	int iCmpSizeOutY,
	bool bSum
)
{	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y >= iCmpSizeOutY) return;
	//---------------------------
	int iOut = y * gridDim.x + blockIdx.x;
	if(blockIdx.x >= iCmpSizeInX) return;
	//-----------------------------------
        if(y > (iCmpSizeOutY / 2)) 
	{	y -= iCmpSizeOutY;
		if(y <= (-iCmpSizeInY / 2)) return;
		else y += iCmpSizeInY;
	}
	else
	{	if(y > (iCmpSizeInY / 2)) return;
	}
	//---------------------------------------
	int iIn = y * iCmpSizeInX + blockIdx.x;
	if(bSum)
	{	gCmpOut[iOut].x += gCmpIn[iIn].x;
		gCmpOut[iOut].y += gCmpIn[iIn].y;
	}
	else
	{	gCmpOut[iOut].x = gCmpIn[iIn].x;
		gCmpOut[iOut].y = gCmpIn[iIn].y;
	}
}

GFourierResize2D::GFourierResize2D(void)
{
}

GFourierResize2D::~GFourierResize2D(void)
{
}

void GFourierResize2D::GetBinnedCmpSize
(	int* piCmpSize,
	float fBin,
	int* piNewSize // cmp size after binning
)
{	piNewSize[0] = piCmpSize[0];
	piNewSize[1] = piCmpSize[1];
	if(fBin == 1) return;
	//-------------------
	int aiImgSize[2] = {0};
	aiImgSize[0] = (piCmpSize[0] - 1) * 2;
	aiImgSize[1] = piCmpSize[1];
	GFourierResize2D::GetBinnedImgSize(aiImgSize, fBin, piNewSize);
	piNewSize[0] = piNewSize[0] / 2 + 1;
	piNewSize[1] = piNewSize[1]; 
}

void GFourierResize2D::GetBinnedImgSize
(	int* piImgSize,
	float fBin,
	int* piNewSize
)
{	piNewSize[0] = piImgSize[0];
	piNewSize[1] = piImgSize[1];
	if(fBin == 1.0f) return;
	//--------------------------
	piNewSize[0] = (int)(piImgSize[0] / fBin);
	piNewSize[1] = (int)(piImgSize[1] / fBin);
	//----------------------------------------
	piNewSize[0] = piNewSize[0] / 2 * 2;
	piNewSize[1] = piNewSize[1] / 2 * 2;
}

float GFourierResize2D::CalcPixSize
(	int* piImgSize,
	float fBin,
	float fPixSize  // before binning
)
{	int aiNewSize[2] = {0};
	GFourierResize2D::GetBinnedImgSize(piImgSize, fBin, aiNewSize);
	float fPixSizeX = piImgSize[0] * fPixSize / aiNewSize[0];
	float fPixSizeY = piImgSize[1] * fPixSize / aiNewSize[1];
	return (fPixSizeX + fPixSizeY) * 0.5f;
}

void GFourierResize2D::GetBinning
(	int* piCmpSize, 
	int* piNewSize,
	float* pfBinning
)
{	pfBinning[0] = 1.0f;
	pfBinning[1] = 1.0f;
	//------------------
	if(piCmpSize[0] != piNewSize[0])
	{	pfBinning[0] = (piCmpSize[0] - 1.0f) / (piCmpSize[0] - 1.0f);
	}
	if(piCmpSize[1] != piNewSize[1])
	{	pfBinning[1] = piCmpSize[1] / (float)piCmpSize[1];
	}
}

void GFourierResize2D::DoIt
( 	hipfftComplex* gCmpIn, 
	int* piSizeIn,
  	hipfftComplex* gCmpOut, 
	int* piSizeOut,
	bool bSum,
	hipStream_t stream
)
{	dim3 aBlockDim(1, 64);
	dim3 aGridDim(piSizeOut[0], 1);
	aGridDim.y = (piSizeOut[1] + aBlockDim.y - 1) / aBlockDim.y;
	//----------------------------------------------------------
	mGResize<<<aGridDim, aBlockDim, 0, stream>>>
	( gCmpIn, piSizeIn[0], piSizeIn[1], 
	  gCmpOut, piSizeOut[1], bSum );	
}

