#include "hip/hip_runtime.h"
#include "CMaUtilInc.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace McAreTomo::MaUtil;

static __global__ void mGResize
( 	hipfftComplex* gCmpIn,
	int iCmpSizeInX,
	int iCmpSizeInY,
  	hipfftComplex* gCmpOut, 
	int iCmpSizeOutY,
	bool bSum
)
{	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(y >= iCmpSizeOutY) return;
	//---------------------------
	int iOut = y * gridDim.x + blockIdx.x;
	if(blockIdx.x >= iCmpSizeInX) return;
	//-----------------------------------
        if(y > (iCmpSizeOutY / 2)) 
	{	y -= iCmpSizeOutY;
		if(y <= (-iCmpSizeInY / 2)) return;
		else y += iCmpSizeInY;
	}
	else
	{	if(y > (iCmpSizeInY / 2)) return;
	}
	//---------------------------------------
	int iIn = y * iCmpSizeInX + blockIdx.x;
	if(bSum)
	{	gCmpOut[iOut].x += gCmpIn[iIn].x;
		gCmpOut[iOut].y += gCmpIn[iIn].y;
	}
	else
	{	gCmpOut[iOut].x = gCmpIn[iIn].x;
		gCmpOut[iOut].y = gCmpIn[iIn].y;
	}
}

GFourierResize2D::GFourierResize2D(void)
{
	m_gfInImg = 0L;
	m_gfOutImg = 0L;
	m_pForwardFFT = new CCufft2D;
	m_pInverseFFT = new CCufft2D;
}

GFourierResize2D::~GFourierResize2D(void)
{
	this->Clean();
	if(m_pForwardFFT != 0L) delete m_pForwardFFT;
	if(m_pInverseFFT != 0L) delete m_pInverseFFT;
}

void GFourierResize2D::GetBinnedCmpSize
(	int* piCmpSize,
	float fBin,
	int* piNewSize // cmp size after binning
)
{	piNewSize[0] = piCmpSize[0];
	piNewSize[1] = piCmpSize[1];
	if(fBin == 1) return;
	//-------------------
	int aiImgSize[2] = {0};
	aiImgSize[0] = (piCmpSize[0] - 1) * 2;
	aiImgSize[1] = piCmpSize[1];
	GFourierResize2D::GetBinnedImgSize(aiImgSize, fBin, piNewSize);
	piNewSize[0] = piNewSize[0] / 2 + 1;
	piNewSize[1] = piNewSize[1]; 
}

void GFourierResize2D::GetBinnedImgSize
(	int* piImgSize,
	float fBin,
	int* piNewSize
)
{	piNewSize[0] = piImgSize[0];
	piNewSize[1] = piImgSize[1];
	if(fBin == 1.0f) return;
	//--------------------------
	piNewSize[0] = (int)(piImgSize[0] / fBin);
	piNewSize[1] = (int)(piImgSize[1] / fBin);
	//----------------------------------------
	piNewSize[0] = piNewSize[0] / 2 * 2;
	piNewSize[1] = piNewSize[1] / 2 * 2;
}

float GFourierResize2D::CalcPixSize
(	int* piImgSize,
	float fBin,
	float fPixSize  // before binning
)
{	int aiNewSize[2] = {0};
	GFourierResize2D::GetBinnedImgSize(piImgSize, fBin, aiNewSize);
	float fPixSizeX = piImgSize[0] * fPixSize / aiNewSize[0];
	float fPixSizeY = piImgSize[1] * fPixSize / aiNewSize[1];
	return (fPixSizeX + fPixSizeY) * 0.5f;
}

void GFourierResize2D::GetBinning
(	int* piCmpSize, 
	int* piNewSize,
	float* pfBinning
)
{	pfBinning[0] = 1.0f;
	pfBinning[1] = 1.0f;
	//------------------
	if(piCmpSize[0] != piNewSize[0])
	{	pfBinning[0] = (piCmpSize[0] - 1.0f) / (piCmpSize[0] - 1.0f);
	}
	if(piCmpSize[1] != piNewSize[1])
	{	pfBinning[1] = piCmpSize[1] / (float)piCmpSize[1];
	}
}

void GFourierResize2D::DoIt
( 	hipfftComplex* gCmpIn, 
	int* piSizeIn,
  	hipfftComplex* gCmpOut, 
	int* piSizeOut,
	bool bSum,
	hipStream_t stream
)
{	dim3 aBlockDim(1, 64);
	dim3 aGridDim(piSizeOut[0], 1);
	aGridDim.y = (piSizeOut[1] + aBlockDim.y - 1) / aBlockDim.y;
	//-----------------
	mGResize<<<aGridDim, aBlockDim, 0, stream>>>
	( gCmpIn, piSizeIn[0], piSizeIn[1], 
	  gCmpOut, piSizeOut[1], bSum );	
}

void GFourierResize2D::Clean(void)
{
	m_pForwardFFT->DestroyPlan();
	m_pInverseFFT->DestroyPlan();
	if(m_gfInImg != 0L) hipFree(m_gfInImg);
	if(m_gfOutImg != 0L) hipFree(m_gfOutImg);
	m_gfInImg = 0L;
	m_gfOutImg = 0L;
}

void GFourierResize2D::Setup
(	int* piInImgSize,
	int* piOutImgSize
)
{	this->Clean();
	//-----------------
	memcpy(m_aiInImgSize, piInImgSize, sizeof(int) * 2);
	memcpy(m_aiOutImgSize, piOutImgSize, sizeof(int) * 2);
	//-----------------
	m_pForwardFFT->CreateForwardPlan(m_aiInImgSize, false);
	m_pInverseFFT->CreateInversePlan(m_aiOutImgSize, false);
	//-----------------
	size_t tBytes = sizeof(float) * m_aiInImgSize[1] *
	   (m_aiInImgSize[0] / 2 + 1) * 2;
	hipMalloc(&m_gfInImg, tBytes);
	//-----------------
	tBytes = sizeof(float) * m_aiOutImgSize[1] *
	   (m_aiOutImgSize[0] / 2 + 1) * 2;
	hipMalloc(&m_gfOutImg, tBytes);
}

void GFourierResize2D::DoIt(float* pfInImg, float* pfOutImg)
{
	CPad2D pad2D;
	pad2D.Pad(pfInImg, m_aiInImgSize, m_gfInImg);
	m_pForwardFFT->Forward(m_gfInImg, true);
	//-----------------
	hipfftComplex* gInCmp = (hipfftComplex*)m_gfInImg;
	hipfftComplex* gOutCmp = (hipfftComplex*)m_gfOutImg;
	//-----------------
	int aiInCmpSize[] = {0, m_aiInImgSize[1]};
	int aiOutCmpSize[] = {0, m_aiOutImgSize[1]};
	aiInCmpSize[0] = m_aiInImgSize[0] / 2 + 1;
	aiOutCmpSize[0] = m_aiOutImgSize[0] / 2 + 1;
	this->DoIt(gInCmp, aiInCmpSize, gOutCmp, aiOutCmpSize, false);
	//-----------------
	m_pInverseFFT->Inverse(gOutCmp);
	int aiOutPadSize[] = {aiOutCmpSize[0] * 2, aiOutCmpSize[1]};
	pad2D.Unpad(m_gfOutImg, aiOutPadSize, pfOutImg);
}
